#include "hip/hip_runtime.h"
#include <SDL_Timer.h>
#include "work.h"
#include "common.h"

__global__
void Work_c(const char *name, WorkFunc func, int32_t count, thread_t &threadpool) {
	hipMemset(&work, 0, sizeof(work));

	work.lock = SDL_CreateMutex(); // is the mutex needed in this case?
	work.name = name;
	work.count = count;
	work.func = func;
	work.index = 0;
	work.percent = -1;

	const uint32_t start = SDL_GetTicks();
	threadpool[threadIdx.x] = Thread_Create(RunFunc, NULL, 0);
	Thread_Wait(threads[threadId.x]);

	const uint32_t end = SDL_GetTicks();

	if (work.name) {
		Com_Print(" %d ms\n", end - start);
	}
}

void dispatchWork(const char *name, WorkFunc func, int32_t count) {
	cudaGetDeviceProp *prop;
	hipGetDeviceProperties ( &prop );
	int numThreads = prop.maxThreadsPerBlock;
	thread_t *threads[thread_count];
	Work_c<<<1, numThreads>>>(*name, func, count, &threads);
	cudaDeviceSynchonize();
}
