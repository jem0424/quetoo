#include "hip/hip_runtime.h"
/*
 * Copyright(c) 1997-2001 id Software, Inc.
 * Copyright(c) 2002 The Quakeforge Project.
 * Copyright(c) 2006 Quetoo.
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "work.h"

__global__ void GetWork(void) {

}

extern "C" {

/**
 * @brief
 */
int32_t WorkCuda(work_t *work) {

	hipInit(0);

	int count;
	hipGetDeviceCount(&count);

	if (count > 0) {

		hipDevice_t device;
		hipDeviceGet(&device, 0);

		int threads;
		hipDeviceGetAttribute(&threads, hipDeviceAttributeMaxThreadsPerBlock, device);

		printf("Let's run some threads.. %d\n", threads);
	}

	return 0;
}

}
