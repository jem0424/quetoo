#include "hip/hip_runtime.h"
/*
 * Copyright(c) 1997-2001 id Software, Inc.
 * Copyright(c) 2002 The Quakeforge Project.
 * Copyright(c) 2006 Quetoo.
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "work.h"

/**
 * @brief CUDA kernel to run the configured WorkFunc for the current CUDA thread.
 */
__global__ void RunWorkFunc(work_t *work) {
	work->func(work->index + threadIdx.x);
}

extern "C" {

/**
 * @brief
 */
int32_t WorkCuda(work_t *work) {

	hipInit(0);

	int count;
	hipGetDeviceCount(&count);

	if (count > 0) {

		hipDevice_t device;
		hipDeviceGet(&device, 0);

		int threads;
		hipDeviceGetAttribute(&threads, hipDeviceAttributeMaxThreadsPerBlock, device);

		while (work->index < work->count) {

			int n = threads;

			if (work->index + n > work->count) {
				n = work->count - work->index;
			}

			RunWorkFunc<<<1, n>>>(work);

			hipCtxSynchronize();

			work->index += n;

			printf("Ran %d threads\n", n);
		}
	}

	return count;
}

}
